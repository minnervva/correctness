#include "hip/hip_runtime.h"
#include <vector>
#include <whip.hpp>

#ifdef USE_MAIN
#include <hiprand.h>

/* ref : https://en.wikipedia.org/wiki/Kahan_summation_algorithm */
template <typename T>
T KahanBabushkaNeumaierSum(const T *data__, const int size__) {
  T sum = (T)0;
  T c = (T)0;
  for (int i = 0; i < size__; i++) {
    T t = sum + data__[i];
    if (std::abs(sum) > std::abs(data__[i]))
      c +=
          (sum - t) +
          data__[i]; // If sum is bigger, low-order digits of input[i] are lost.
    else
      c += (data__[i] - t) + sum; // Else low-order digits of sum are lost.
    sum = t;
  }
  return sum + c;
}

template <typename T> T KahanBabushkaNeumaierSum(const std::vector<T> &data__) {
  T sum = (T)0;
  T c = (T)0;
  for (auto &elm_ : data__) {
    T t = sum + elm_;
    if (std::abs(sum) > std::abs(elm_))
      c += (sum - t) +
           elm_; // If sum is bigger, low-order digits of input[i] are lost.
    else
      c += (elm_ - t) + sum; // Else low-order digits of sum are lost.
    sum = t;
  }
  return sum + c;
}
#else
#include "utils.hpp"
#endif

#ifdef REDUCE_USE_CUDA
#include <hipcub/hipcub.hpp>
#else
#include <hipcub/hipcub.hpp>
#endif
/* ---------------------------------------------------------------------------
 */
/* Taken from the cuda samples sdk */
/* ----------------------------------------------------------------------------*/

template <typename T> struct SharedMemory {
  __device__ inline operator T *() {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

// specialize for double to avoid unaligned memory
// access compile errors
template <> struct SharedMemory<double> {
  __device__ inline operator double *() {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }

  __device__ inline operator const double *() const {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }
};

template <class T, int blockSize, bool shuffle__ = false>
__device__ __inline__ void warpReduce(T *sdata, T &mySum) {
#ifdef REDUCE_USE_CUDA
  if (shuffle__) { // shuffle
    if (threadIdx.x < warpSize) {
      for (int offset = warpSize / 2; offset > 0; offset /= 2)
        mySum += __shfl_down_sync(0xffffffff, mySum, offset);
    }
    return;
  } else {
    // fully unroll reduction within a single warp
  if (threadIdx.x < 16)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x < 8)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x < 4)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x < 2)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x < 1)
    mySum = mySum + sdata[threadIdx.x + 1];
  __syncthreads();
  }
#else
  // hip only supports __shfl_down_sync starting from ROCM 6.3
  // so we stick shared memory reduction on AMD hardware for the time being
  // __syncwarp() is not supported by hip.
  // fully unroll reduction within a single warp
  if (threadIdx.x < 16)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x < 8)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x < 4)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x < 2)
    sdata[threadIdx.x] = mySum = mySum + sdata[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x < 1)
    mySum = mySum + sdata[threadIdx.x + 1];
  
#endif
  return;
}
/* it can also be done with shuffle and cooperative groups. cooperative groups
   is supported on AMD with ROCM 6 I think. Does not offer anything special
   except compacity.

   The block reduction can also be implemented with the warpReduction only but
   it will change the order of the operations and we will have to be careful
   with the warp size difference between NVIDIA and AMD GPU if we want a cross
   platform code.


*/

template <class T, int blockSize, bool shuffle__>
__device__ __inline__ void blockReduce(T *sdata, T &mySum,
                                       const unsigned int tid) {
  // each thread puts its local sum into shared memory
  sdata[tid] = mySum;
  __syncthreads();

  // do reduction in shared mem
  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] = mySum = mySum + sdata[tid + 256];
  }
  __syncthreads();

  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] = mySum = mySum + sdata[tid + 128];
  }
  __syncthreads();

  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] = mySum = mySum + sdata[tid + 64];
  }
  __syncthreads();

  if ((blockSize >= 64) && (tid < 32)) {
    sdata[tid] = mySum = mySum + sdata[tid + 32];
  }
  __syncthreads();

  // apply warp reduction for the first warp of the thread group.
  warpReduce<T, blockSize, shuffle__>(sdata, mySum);
  // __syncthreads();
}
/*
  All the code is taken From Mark Harris (NVIDIA) original work on
  reduction. It is probably a bit outdated nowadays as the block reduction
  can be done with specific hardware instructions. This code
  is generic and will work equally well on AMD and NVIDIA GPUs

  This version adds multiple elements per thread sequentially.  This reduces
  the overall cost of the algorithm while keeping the work complexity O(n) and
  the step complexity O(log n). (Brent's Theorem optimization)

  Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
  In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
  If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/

template <class T, unsigned int blockSize, bool shuffle__ = false,
          bool single_round_ = false, bool nIsPow2, bool deterministic = true>
__global__ void reduce_gpu(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  T *sdata = SharedMemory<T>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
  unsigned int gridSize = blockSize * 2 * gridDim.x;
  // use the end of scratch memory to store how many block are already done.

  unsigned int *retirementCount = (unsigned int *)(g_odata + gridDim.x + 1);
  T mySum = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n) {
    mySum += g_idata[i];

    // ensure we don't read out of bounds -- this is optimized away for powerOf2
    // sized arrays
    if (nIsPow2 || i + blockSize < n)
      mySum += g_idata[i + blockSize];

    i += gridSize;
  }

  blockReduce<T, blockSize, shuffle__>(sdata, mySum, tid);

  if (deterministic) {
    // write result for this block to global mem. We do not optimize away the
    // global write when we use the single pass algorithm with determinism
    // because it would otherwise make the code non deterministic which is
    // something we want to avoid at all costs.
    if (tid == 0)
      g_odata[blockIdx.x] = mySum;
    // We have the option to let the last block do the final reduction instead
    // of either calling the function once more or copy the partial results to
    // CPU and do the final reduction on CPU. Both ways are deterministic but
    // will lead to slightly different answers because of the order of the
    // operations.

    // On CPU we use Kahan while on GPU we use the tree reduction again. Which
    // one is faster is also interesting.

    if (single_round_) {
      if (gridDim.x > 1) {
        __shared__ bool amLast;

        // wait until all outstanding memory instructions in this thread are
        // Finished
        __threadfence();

        // Thread 0 takes a ticket
        if (tid == 0) {
          unsigned int ticket = atomicInc(retirementCount, gridDim.x);
          // If the ticket ID is equal to the number of blocks, we are the last
          // block!
          amLast = (ticket == gridDim.x - 1);
        }

        __syncthreads();

        // The last block sums the results of all other blocks
        if (amLast) {
          int i = tid;
          mySum = (T)0;

          while (i < gridDim.x) {
            mySum += g_odata[i];
            i += blockSize;
          }

          blockReduce<T, blockSize, shuffle__>(sdata, mySum, tid);

          if (tid == 0) {
            g_odata[0] = mySum;
          }
        }
      }
    }
  } else {
    if (tid == 0)
      atomicAdd(&g_odata[0], mySum);
    return;
  }
}

/* ----------------------------------------------------------------------------------
 */

template <typename T, bool shuffle__ = false, bool single_round_ = false,
          bool deterministic = true>
void reduce_step(whip::stream_t stream, int size, int threads, int blocks,
                 T *d_idata, T *d_odata) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

  switch (threads) {
  case 512:
    reduce_gpu<T, 512, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 256:
    reduce_gpu<T, 256, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 128:
    reduce_gpu<T, 128, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 64:
    reduce_gpu<T, 64, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 32:
    reduce_gpu<T, 32, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;
  case 16:
    reduce_gpu<T, 16, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 8:
    reduce_gpu<T, 8, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 4:
    reduce_gpu<T, 4, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 2:
    reduce_gpu<T, 2, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;

  case 1:
    reduce_gpu<T, 1, shuffle__, single_round_, false, deterministic>
        <<<dimGrid, dimBlock, smemSize, stream>>>(d_idata, d_odata, size);
    break;
  }
}

int get_block_size(const int size__) {
  int block_size = 32;
  if (size__ > 64) {
    block_size = 128;
  }
  if (size__ > 128) {
    block_size = 256;
  }
  if (size__ > 256) {
    block_size = 512;
  }
  return block_size;
}

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the reduction
// We set threads / block to the minimum of maxThreads and n/2.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks,
                            int &threads) {
  if (n == 1) {
    threads = 1;
    blocks = 1;
  } else {
    threads = (n < maxThreads * 2) ? nextPow2(n / 2) : maxThreads;
    blocks = max(1, n / (threads * 2));
  }

  blocks = min(maxBlocks, blocks);
}

template <class T> __global__ void reduce_atomic_cuda(const T*__restrict__ data__, const size_t size__, T *scratch__)
{
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= size__)
    return;
  atomicAdd(scratch__, data__[tid]);
}

template <class T> T reduce_atomic(whip::stream_t stream__, const int size__, T*data__, T*scratch__)
{
  const int block_size_ = 128;
  const int grid_size_ = (size__ + block_size_ - 1) / block_size_;
  T result;
  whip::memset_async(scratch__, 0, sizeof(T), stream__);
  reduce_atomic_cuda<<<grid_size_, block_size_, 0, stream__>>>(data__, size__, scratch__);
  whip::memcpy_async(&result, scratch__, sizeof(T), whip::memcpy_device_to_host, stream__);
  whip::stream_synchronize(stream__);
  return result;
}

template <typename T, bool shuffle__ = false, bool deterministic = true>
T reduce(whip::stream_t stream__, const bool single_round__,
         const int block_size__, const int grid_size__, const int size__,
         T *data__, T *scratch__) {
  T *in, *out;
  int grid_size = 1;
  int block_size = 1;
  in = data__;
  out = scratch__;
  double result = 0.0;

  block_size = block_size__;
  if (block_size__ < 64)
    block_size = 64;

  if (block_size__ > 512)
    block_size = 512;

  grid_size = (size__ + 2 * block_size - 1) / (2 * block_size);
  if ((grid_size__ > 0) && (grid_size__ < grid_size))
    grid_size = grid_size__;

  if (deterministic) {
    // number of threads per block
    // size of the block grid. The reduce algorithm add at least two elements
    // per thread. so the block grid should NOT be size__/block_size but
    // size__/ 2 * block_size__).
    std::vector<T> tmp(grid_size, 0.0);
    if (single_round__) {
      whip::memset_async(out + grid_size + 1, 0, sizeof(T), stream__);
      reduce_step<T, shuffle__, true, true>(stream__, size__, block_size,
                                            grid_size, in, out);
      whip::memcpy_async(&result, out, sizeof(T), whip::memcpy_device_to_host,
                         stream__);
      whip::stream_synchronize(stream__);
    } else {
      reduce_step<T, shuffle__, false, true>(stream__, size__, block_size,
                                             grid_size, in, out);
      whip::memcpy_async(tmp.data(), out, sizeof(T) * grid_size,
                         whip::memcpy_device_to_host, stream__);

      // we do the final stage on CPU but we make sure the summation is
      // deterministic. It is also possible to get the same answer on GPU only
      // if we iterate the same stage iteratively
      whip::stream_synchronize(stream__);
      result = KahanBabushkaNeumaierSum<T>(tmp);
    }
  } else {
    whip::memset_async(out, 0, sizeof(T), stream__);
    reduce_step<T, shuffle__, false, false>(stream__, size__, block_size,
                                            grid_size, in, out);
    whip::memcpy_async(&result, out, sizeof(T), whip::memcpy_device_to_host,
                       stream__);
    whip::stream_synchronize(stream__);
  }

  return result;
}

template <typename T>
T cub_reduce(whip::stream_t stream__, size_t &temp_storage_bytes,
             T *temp_storage, const size_t size__, T *data__, T *scratch__) {
  T res__ = 0;
#ifdef REDUCE_USE_CUDA
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, data__,
                         static_cast<double *>(scratch__), size__, stream__);
#else
  hipcub::DeviceReduce::Sum(temp_storage, temp_storage_bytes, data__,
                            static_cast<double *>(scratch__), size__, stream__);
#endif

  if (scratch__ != NULL)
    whip::memcpy_async(&res__, scratch__, sizeof(T),
                       whip::memcpy_device_to_host, stream__);
  whip::stream_synchronize(stream__);
  return res__;
}

template <typename T>
T reduce_step_det(whip::stream_t stream__, const int fixed_block__,
                  const int grid_size__, const int size__, T *data__,
                  T *scratch__) {
  return reduce<T, false, true>(stream__, false, fixed_block__, grid_size__,
                                size__, data__, scratch__);
}

template <typename T>
T reduce_step_det_shuffle(whip::stream_t stream__, const int fixed_block__,
                          const int grid_size__, const int size__, T *data__,
                          T *scratch__) {
  return reduce<T, true, true>(stream__, false, fixed_block__, grid_size__,
                               size__, data__, scratch__);
}

template <typename T>
T reduce_step_non_det_shuffle(whip::stream_t stream__, const int fixed_block__,
                              const int grid_size__, const int size__,
                              T *data__, T *scratch__) {
  return reduce<T, true, true>(stream__, false, fixed_block__, grid_size__,
                               size__, data__, scratch__);
}

template <typename T>
T reduce_step_non_det(whip::stream_t stream__, const int fixed_block__,
                      const int grid_size__, const int size__, T *data__,
                      T *scratch__) {
  return reduce<T, false, false>(stream__, false, fixed_block__, grid_size__,
                                 size__, data__, scratch__);
}

template <typename T>
T reduce_step_det_single_round(whip::stream_t stream__, const int fixed_block__,
                               const int grid_size__, const int size__,
                               T *data__, T *scratch__) {
  return reduce<T, false, true>(stream__, true, fixed_block__, grid_size__,
                                size__, data__, scratch__);
}

template <typename T>
T reduce_step_det_shuffle_single_round(whip::stream_t stream__,
                                       const int fixed_block__,
                                       const int grid_size__, const int size__,
                                       T *data__, T *scratch__) {
  return reduce<T, true, true>(stream__, true, fixed_block__, grid_size__,
                               size__, data__, scratch__);
}

template double reduce_step_det<double>(whip::stream_t stream__,
                                        const int fixed_block__,
                                        const int grid_size__, const int size__,
                                        double *data__, double *scratch__);
template double reduce_step_det_shuffle<double>(
    whip::stream_t stream__, const int fixed_block__, const int grid_size__,
    const int size__, double *data__, double *scratch__);
template double reduce_step_det_single_round<double>(
    whip::stream_t stream__, const int fixed_block__, const int grid_size__,
    const int size__, double *data__, double *scratch__);
template double reduce_step_det_shuffle_single_round<double>(
    whip::stream_t stream__, const int fixed_block__, const int grid_size__,
    const int size__, double *data__, double *scratch__);
template double reduce_step_non_det<double>(whip::stream_t stream__,
                                            const int fixed_block__,
                                            const int grid_size__,
                                            const int size__, double *data__,
                                            double *scratch__);
template double reduce_step_non_det_shuffle<double>(
    whip::stream_t stream__, const int fixed_block__, const int grid_size__,
    const int size__, double *data__, double *scratch__);
template double cub_reduce(whip::stream_t stream__, size_t &temp_storage_bytes,
                           double *temp_storage, const size_t size__,
                           double *data__, double *scratch__);

template double reduce_atomic<double>(whip::stream_t stream__, const int size__, double*data__, double*scratch__);

#ifdef USE_MAIN
int main(int argc, char **argv) {
  double *table = nullptr;
  double *scratch = nullptr;
  double *temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  int number_of_fp_elements = 1000000;
  hiprandGenerator_t gen_;
  whip::stream_t stream_;

  whip::stream_create(&stream_);
  hiprandCreateGenerator(&gen_, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen_, 0x234abe);

  whip::malloc(&table, sizeof(double) * number_of_fp_elements);
  whip::malloc(&scratch, sizeof(double) * number_of_fp_elements);

  hiprandGenerateUniformDouble(gen_, table, number_of_fp_elements);
  // we ignore the result
  cub_reduce(stream_, temp_storage_bytes, temp_storage, number_of_fp_elements,
             table, scratch);

  whip::malloc(&temp_storage, sizeof(double) * temp_storage_bytes);
  for (int round = 0; round < 10; round++) {
    // reduction using cub
    double res1 = cub_reduce(stream_, temp_storage_bytes, temp_storage,
                             number_of_fp_elements, table, scratch);
    // reduction using the two stages reduction
    double res2 = reduce_step_det<double>(
        stream_, 512, -1, number_of_fp_elements, table, scratch);
    // use the thread_fence instruction and a counter. Check the case
    // deterministic +  single
    double res3 = reduce_step_det_single_round<double>(
        stream_, 512, -1, number_of_fp_elements, table, scratch);
    uint64_t resi1;
    uint64_t resi2;
    uint64_t resi3;

    memcpy(&resi1, &res1, sizeof(double));
    memcpy(&resi2, &res2, sizeof(double));
    memcpy(&resi3, &res3, sizeof(double));
    std::cout << "Results : supposed to be deterministic: " << resi3
              << " Deterministic two stages: " << resi2;
    std::cout << " CUb version: " << resi1 << std::endl;
  }

  whip::stream_destroy(stream_);
  whip::free(table);
  whip::free(scratch);
  whip::free(temp_storage);
  hiprandDestroyGenerator(gen_);

  return 0;
}
#endif
